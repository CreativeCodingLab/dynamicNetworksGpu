#include "hip/hip_runtime.h"
/*
*
* Program: Pearson Correlatrion Coefficient computation.
* Author: Andrea Purgato
* Version: counter occurences version.
*
* File: DeviceReader.cu
* Description: this file support the program with some functions that are related to the performance of the GPU devices.
*
*/


#pragma once

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include "Logger.cpp"

/*
	Struct hat contains the relevant GPU info.
*/
struct device{
	char *name;
	size_t globalMem;
	int warpSize;
	int maxThreadPerBlock;
	int maxBlockSize[3];
	int maxGridSize[3];
	int cuncurrentKernels;
	int registerPerBlock;
};

/*
	Struct used to store the performance.
*/
struct gpuPerformance{
	gpuPerformance() : millisecAvgVariance(0), millisecAvgCovariance(0), millisecAvgCorrelation(0), millisecTotVariance(0), millisecTotCovariance(0), millisecTotCorrelation(0){};
	float millisecAvgVariance;
	float millisecAvgCovariance;
	float millisecAvgCorrelation;
	float millisecTotVariance;
	float millisecTotCovariance;
	float millisecTotCorrelation;
	float bwVariance;
	float bwCovariance;
	float bwCorrelation;
};

struct cpuPerformance{
	cpuPerformance() : exeTime(0), exeSaving(0), waitingTime(0){};
	int exeTime;
	int exeSaving;
	int waitingTime;
};

/*
	Function that get back the number of devices.
*/
int getDeviceNumber() {

	/// Query the number of devices.
	int d;
	hipGetDeviceCount(&d);

	//return d;
	return 1;
}

/*
	Function that create the arrays of gpus properties.
*/
device* getDeviceProp() {

	int n = getDeviceNumber();

	device *d = (device*)malloc(n * sizeof(device));
	for (int i = 0; i < n; i++) {

		/// Query the properties.
		hipDeviceProp_t p;
		hipGetDeviceProperties(&p, i);

		d[i].name = p.name;
		d[i].globalMem = p.totalGlobalMem;
		d[i].maxThreadPerBlock = p.maxThreadsPerBlock;

		d[i].maxBlockSize[0] = p.maxThreadsDim[0];
		d[i].maxBlockSize[1] = p.maxThreadsDim[1];
		d[i].maxBlockSize[2] = p.maxThreadsDim[2];

		d[i].maxGridSize[0] = p.maxGridSize[0];
		d[i].maxGridSize[1] = p.maxGridSize[1];
		d[i].maxGridSize[2] = p.maxGridSize[2];

		d[i].cuncurrentKernels = p.concurrentKernels;
		d[i].registerPerBlock = p.regsPerBlock;
		d[i].warpSize = p.warpSize;

		log("Device " + std::to_string(i) + " " + p.name);
		//log("Device memory:" + std::to_string(p.totalGlobalMem / (1024 * 1024)) + " GB");
		//log("Warp size: " + std::to_string(p.warpSize));
		//log("Register per block: " + std::to_string(p.regsPerBlock));
		//log("Register per multiprocesssor: " + std::to_string(p.regsPerMultiprocessor));

	}

	return d;
}

/*
	Function that update the performance measure.
*/
void updatePerformance(gpuPerformance* perf, float millisec, int N, int stuff){

	if (stuff == 1){

		if (perf->millisecAvgVariance == 0)
			perf->millisecAvgVariance = millisec;
		else
			perf->millisecAvgVariance = (perf->millisecAvgCovariance + millisec) / 2;

		perf->millisecTotVariance = perf->millisecTotVariance + millisec;
		perf->bwVariance = (N * sizeof(int) * 4 + N * sizeof(float)) / millisec / 1e6;

	}

	if (stuff == 2){

		if (perf->millisecAvgCovariance == 0)
			perf->millisecAvgCovariance = millisec;
		else
			perf->millisecAvgCovariance = (perf->millisecAvgCovariance + millisec) / 2;

		perf->millisecTotCovariance = perf->millisecTotCovariance + millisec;
		perf->bwCovariance = (N * sizeof(int) * 4 + N * sizeof(float)) / millisec / 1e6;

	}

	if( stuff == 3) {

		if (perf->millisecAvgCorrelation == 0)
			perf->millisecAvgCorrelation = millisec;
		else
			perf->millisecAvgCorrelation = (perf->millisecAvgCorrelation + millisec) / 2;

		perf->millisecTotCorrelation = perf->millisecTotCorrelation + millisec;
		perf->bwCorrelation = (N * sizeof(float) * 4) / millisec / 1e6;

	}

}

/*
	Function used to save the performance.
*/
void savePerformance(int gpuNumber, int window, gpuPerformance* perf, cpuPerformance* cpuPerf, int pixels){

	std::string RES_FOLDER = "output/";
	std::string fileName = "N_" + std::to_string(pixels) + "_W_" + std::to_string(window) + "_performance.txt";

	/// Open the performance file.
	std::ofstream oFile(RES_FOLDER + fileName);

	for (int i = 0; i < gpuNumber; i++){

		oFile << "Device " + std::to_string(i) + " Avg Variance Time," + std::to_string(perf[i].millisecAvgVariance) << "\n";
		oFile << "Device " + std::to_string(i) + " Tot Variance Time," + std::to_string(perf[i].millisecTotVariance) << "\n";

		oFile << "Device " + std::to_string(i) + " Avg Covariance Time," + std::to_string(perf[i].millisecAvgCovariance) << "\n";
		oFile << "Device " + std::to_string(i) + " Tot Covariance Time," + std::to_string(perf[i].millisecTotCovariance) << "\n";

		oFile << "Device " + std::to_string(i) + " Avg Correlation Time," + std::to_string(perf[i].millisecAvgCorrelation) << "\n";
		oFile << "Device " + std::to_string(i) + " Tot Correlation Time," + std::to_string(perf[i].millisecTotCorrelation) << "\n";

		oFile << "Device " + std::to_string(i) + " BW Variance," + std::to_string(perf[i].bwVariance) << "\n";
		oFile << "Device " + std::to_string(i) + " BW Covariance," + std::to_string(perf[i].bwCovariance) << "\n";
		oFile << "Device " + std::to_string(i) + " BW Correlation," + std::to_string(perf[i].bwCorrelation) << "\n";

	}

	oFile << "CPU Tot Execution Time," + std::to_string((cpuPerf->exeTime) * 1000) << "\n";
	oFile << "CPU Tot Saving Time," + std::to_string((cpuPerf->exeSaving) * 1000) << "\n";
	oFile << "CPU Tot Waiting Time," + std::to_string((cpuPerf->waitingTime) * 1000) << "\n";

	/// Close the file.
	oFile.close();

}
